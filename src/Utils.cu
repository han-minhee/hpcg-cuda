#include "Utils.cuh"

int numSM;
int numSP;

hipDeviceProp_t deviceProp;
int threadsPerBlock;
int regsPerBlock;
int multiProcessorCount;
int deviceWarpSize;

int getWarpSize(void) { return deviceWarpSize; }
int getThreadsPerBlock(void) { return threadsPerBlock; }
int getRegsPerBlock(void) { return regsPerBlock; }
int getMultiProcessorCount(void) { return multiProcessorCount; }

int gpuAssert(hipError_t code) {
  if (code != hipSuccess) {
    printf("CUDA Error (%d): %s\n", code, hipGetErrorString(code));
    return -1;
  } else {
    return 0;
  }
}

int gpuCheckError(void) { return gpuAssert(hipPeekAtLastError()); }

int initDevice(void) {
  
  if (hipSuccess != hipGetDeviceProperties(&deviceProp, 0)) {
    printf("CUDA device property request failed\n");
    return -1;
  }

  deviceWarpSize = deviceProp.warpSize;
  regsPerBlock = deviceProp.regsPerBlock;
  multiProcessorCount = deviceProp.multiProcessorCount;
  printf("\n\n\n");
  printf("=== CUDA Platform Information ===\n");
  printf("deviceName : %s\n", deviceProp.name);
  printf("deviceWarpSize : %d\n", deviceWarpSize);
  printf("regsPerBlock : %d\n", regsPerBlock);
  printf("multiProcessorCount : %d\n", multiProcessorCount);
  printf("maxThreadsPerBlock : %d\n", deviceProp.maxThreadsPerBlock);

  printf("=== CUDA Platform Information ===\n\n\n");

  return 0;
}