#include "VectorInside.cuh"
#include <hip/hip_runtime.h>

 void CudaInitializeVectorInside(Vector &v, local_int_t localLength) {
  v.localLength = localLength;
  v.optimizationData = 0;
  hipMalloc((void **)&v.d_values, localLength * sizeof(double));
}

 void CudaZeroVectorInside(Vector &v) {
  hipMemset((void **)&v.d_values, 0.0, v.localLength * sizeof(double));
}

 void CudaScaleVectorValueInside(Vector &v, local_int_t index, double value) {
  assert(index >= 0 && index < v.localLength);
  ScaleVectorValue(v, index, value);
  vectorMemcpyFromHostToDevice(v);
  return;
}

// TODO: chage to Curand later
 void CudaFillRandomVectorInside(Vector &v) {
  local_int_t localLength = v.localLength;
  double *vv = v.values;
  for (int i = 0; i < localLength; ++i)
    vv[i] = rand() / (double)(RAND_MAX) + 1.0;

  hipMemcpy(v.d_values, vv, localLength * sizeof(double),
             hipMemcpyHostToDevice);
}

 void CudaCopyVectorInside(const Vector &v, Vector &w) {
  hipMemcpy(w.d_values, v.d_values, v.localLength * sizeof(double),
             hipMemcpyDeviceToDevice);
}

 void CudaDeleteVectorInside(Vector &v) {
  hipFree(v.d_values);
  v.localLength = 0;
}
