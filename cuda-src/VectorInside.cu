#include "VectorInside.cuh"
#include <hip/hip_runtime.h>
#include <thrust/fill.h>
#include <thrust/device_ptr.h>
#include <vector>


void CudaInitializeVectorInside(Vector &v, local_int_t localLength) {
  v.localLength = localLength;
  v.optimizationData = 0;
  hipMalloc((void **)&v.d_values, localLength * sizeof(double));
}

void CudaZeroVectorInside(Vector &v) {
  // local_int_t localLength = v.localLength;
  // hipFree(v.d_values);
  // hipMalloc((void **)&v.d_values, v.localLength * sizeof(double));
  thrust::device_ptr<double> dev_ptr(v.d_values);
  thrust::fill(dev_ptr, dev_ptr + v.localLength, 0);
  // hipMemset(v.d_values, 0, v.localLength * sizeof(double));
    // hipMemset((void **)&v.d_values, 0, v.localLength * sizeof(double));
      // hipMemset((void **)&v.d_values, 0.0f, v.localLength * sizeof(double));
}

void CudaScaleVectorValueInside(Vector &v, local_int_t index, double value) {
  assert(index >= 0 && index < v.localLength);
  double *vv = v.values;
  vv[index] *= value;
  vectorMemcpyFromHostToDevice(v);
  return;
}

// TODO: chage to Curand later
void CudaFillRandomVectorInside(Vector &v) {
  std::vector<double> rng(v.localLength);
  for(int i = 0; i < v.localLength; ++i)
  {
    rng[i] = rand() / (double)(RAND_MAX) + 1.0;
  }

  hipMemcpy(v.d_values,
                      rng.data(),
                      sizeof(double) * v.localLength,
                      hipMemcpyHostToDevice);
}

void CudaCopyVectorInside(const Vector &v, Vector &w) {
  hipMemcpy(w.d_values, v.d_values, v.localLength * sizeof(double),
             hipMemcpyDeviceToDevice);
}

void CudaDeleteVectorInside(Vector &v) {
  hipFree(v.d_values);
  v.localLength = 0;
}
