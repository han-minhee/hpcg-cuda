#include "hip/hip_runtime.h"
#include "ComputeDotProduct_cuda.cuh"

// notice: originally, for the use of MPI, there should be result,
// time_allreduce variable, but currently omitted. instead of changing a value,
// it returns a double var.

__global__ void kernelDotProduct(int n, double *xv, double *yv,
                                 double *local_results, int deviceWarpSize) {
  int localIndex;
  int elemsPerThreads = deviceWarpSize;

  int globalIndex = blockDim.x * blockIdx.x + threadIdx.x;
  globalIndex *= elemsPerThreads;
  if (globalIndex + elemsPerThreads >= n)
    return;

  for (localIndex = globalIndex; localIndex < globalIndex + elemsPerThreads;
       localIndex++) {
    local_results[localIndex] = xv[localIndex] * yv[localIndex];
  }
}

int ComputeDotProduct_cuda(const local_int_t n, const Vector &x,
                           const Vector &y, double &result,
                           double &time_allreduce) {
  assert(x.localLength >= n); // Test vector lengths
  assert(y.localLength >= n);

  double *xv = x.values;
  double *yv = y.values;
  double *local_results = new double[n]();

  double *xv_d;
  double *yv_d;
  double *local_results_d;

  size_t n_size = n * sizeof(double);

  result = 0.0;

  hipMalloc((void **)&xv_d, n_size);
  if (gpuCheckError() == -1) {
    return -1;
  }

  hipMalloc((void **)&yv_d, n_size);

  if (gpuCheckError() == -1) {
    return -1;
  }

  hipMalloc((void **)&local_results_d, n_size);

  if (gpuCheckError() == -1) {
    return -1;
  }

  hipMemcpy(xv_d, xv, n_size, hipMemcpyHostToDevice);
  if (gpuCheckError() == -1) {
    return -1;
  }

  hipMemcpy(yv_d, yv, n_size, hipMemcpyHostToDevice);
  if (gpuCheckError() == -1) {
    return -1;
  }

  hipMemcpy(local_results_d, local_results, n_size, hipMemcpyHostToDevice);
  if (gpuCheckError() == -1) {
    return -1;
  }

  hipDeviceSynchronize();

  size_t deviceWarpSize = 32;
  int numBlocks = (n + deviceWarpSize - 1) / deviceWarpSize;
  // kernelDotProduct<<<numBlocks, deviceWarpSize>>>(
  //    n, xv_d, yv_d, local_results_d, deviceWarpSize);

  hipblasHandle_t h;
  hipblasCreate(&h);
  hipblasSetPointerMode(h, HIPBLAS_POINTER_MODE_HOST);

  if (gpuCheckError() == -1) {
    return -1;
  }

  hipblasDdot(h, n, xv_d, 1, yv_d, 1, local_results_d);

  hipDeviceSynchronize();

  if (gpuCheckError() == -1) {
    return -1;
  }

  hipMemcpy(local_results, local_results_d, n_size, hipMemcpyDeviceToHost);
  if (gpuCheckError() == -1) {
    return -1;
  }
  hipDeviceSynchronize();

#ifndef HPCG_NO_OPENMP
#pragma omp parallel for reduction(+ : result)
#endif
  for (int i = 0; i < n; i++) {
    result += local_results[i];
  }

  hipFree(xv_d);
  hipFree(yv_d);
  hipFree(local_results_d);
  free(local_results);

  return 0;
}

int ComputeDotProduct_ref_cuda(const local_int_t n, const Vector &x,
                               const Vector &y, double &result,
                               double &time_allreduce) {
  assert(x.localLength >= n); // Test vector lengths
  assert(y.localLength >= n);

  double local_result = 0.0;
  double *xv = x.values;
  double *yv = y.values;
  if (yv == xv) {
#ifndef HPCG_NO_OPENMP
#pragma omp parallel for reduction(+ : local_result)
#endif
    for (local_int_t i = 0; i < n; i++)
      local_result += xv[i] * xv[i];
  } else {
#ifndef HPCG_NO_OPENMP
#pragma omp parallel for reduction(+ : local_result)
#endif
    for (local_int_t i = 0; i < n; i++)
      local_result += xv[i] * yv[i];
  }

#ifndef HPCG_NO_MPI
  // Use MPI's reduce function to collect all partial sums
  double t0 = mytimer();
  double global_result = 0.0;
  MPI_Allreduce(&local_result, &global_result, 1, MPI_DOUBLE, MPI_SUM,
                MPI_COMM_WORLD);
  result = global_result;
  time_allreduce += mytimer() - t0;
#else
  time_allreduce += 0.0;
  result = local_result;
#endif

  return 0;
}