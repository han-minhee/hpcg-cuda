#include "hip/hip_runtime.h"
#include "ComputeDotProduct_cuda.cuh"

// notice: originally, for the use of MPI, there should be result,
// time_allreduce variable, but currently omitted. instead of changing a value,
// it returns a double var.
__global__ void kernelDotProduct(int n, double *xv, double *yv,
                                   double& result, int deviceWarpSize) {
  int localIndex;
  int elemsPerThreads = deviceWarpSize;

  int globalIndex = blockDim.x * blockIdx.x + threadIdx.x;
  globalIndex *= elemsPerThreads;
  if (globalIndex > n)
    return;

  double localResult = 0.0;

  for (localIndex = globalIndex; localIndex < globalIndex + elemsPerThreads;
       localIndex++) {
    localResult += xv[localIndex] * yv[localIndex];
  }

  result = localResult;
}

int ComputeDotProduct_cuda(const local_int_t n, const Vector &x,
                           const Vector &y, double &result,
                           double &time_allreduce) {
  assert(x.localLength >= n); // Test vector lengths
  assert(y.localLength >= n);

  double *xv = x.values;
  double *yv = y.values;
  double *xv_d;
  double *yv_d;

  size_t n_size = n * sizeof(double);
  hipMalloc(&xv_d, n_size);
  hipMalloc(&yv_d, n_size);

  hipMemcpy(xv_d, xv, n_size, hipMemcpyHostToDevice);
  hipMemcpy(yv_d, yv, n_size, hipMemcpyHostToDevice);
  size_t deviceWarpSize = 32;


  int numBlocks = (n + deviceWarpSize - 1) / deviceWarpSize;
  kernelDotProduct<<<numBlocks, deviceWarpSize>>>(n, xv_d, yv_d, result, deviceWarpSize);

  if (gpuAssert(hipPeekAtLastError()) == -1) {
    return -1;
  }
  if (gpuAssert(hipDeviceSynchronize()) == -1) {
    return -1;
  }
}
