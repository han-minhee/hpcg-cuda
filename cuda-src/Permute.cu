#include "hip/hip_runtime.h"
#include "Permute.cuh"
#include "Utils.cuh"

#include <hip/hip_runtime.h>

#define LAUNCH_PERM_COLS(blockSizeX, blockSizeY)                               \
  kernel_perm_cols<blockSizeX, blockSizeY>                                     \
      <<<dim3((A.localNumberOfRows - 1) / blockSizeY + 1),                     \
         dim3(blockSizeX, blockSizeY)>>>(                                      \
          A.localNumberOfRows, A.localNumberOfColumns,                         \
          A.numberOfNonzerosPerRow, A.perm, A.d_mtxIndL, A.d_matrixValues)

template <unsigned int BLOCKSIZE>
__launch_bounds__(BLOCKSIZE) __global__
    void kernel_permute_ell_rows(local_int_t m, local_int_t p,
                                 const local_int_t *tmp_cols,
                                 const double *tmp_vals,
                                 const local_int_t *perm,
                                 local_int_t *ell_col_ind, double *ell_val) {
  local_int_t row = blockIdx.x * BLOCKSIZE + threadIdx.x;

  if (row >= m) {
    return;
  }

  local_int_t idx = p * m + perm[row];
  local_int_t col = tmp_cols[row];

  ell_col_ind[idx] = col;
  ell_val[idx] = tmp_vals[row];
}

__device__ void swap(local_int_t &key, double &val, int mask, int dir) {

  // 32 is for CUDA, temporarily
  local_int_t key1 = __shfl_xor(key, mask);
  __syncthreads();
  //__shfl_xor_sync()(key, mask);
  double val1 = __shfl_xor(val, mask);
  //__shfl_xor_sync(val, mask);
  __syncthreads();
  if (key < key1 == dir) {
    key = key1;
    val = val1;
  }
}

__device__ int get_bit(int x, int i) { return (x >> i) & 1; }

template <unsigned int BLOCKSIZEX, unsigned int BLOCKSIZEY>
__launch_bounds__(BLOCKSIZEX *BLOCKSIZEY) __global__
    void kernel_perm_cols(local_int_t m, local_int_t n,
                          local_int_t nonzerosPerRow, const local_int_t *perm,
                          local_int_t *mtxIndL, double *matrixValues) {
  local_int_t row = blockIdx.x * BLOCKSIZEY + threadIdx.y;
  local_int_t idx = row * nonzerosPerRow + threadIdx.x;
  local_int_t key = n;
  double val = 0.0;

  if (threadIdx.x < nonzerosPerRow && row < m) {
    local_int_t col = mtxIndL[idx];
    val = matrixValues[idx];

    if (col >= 0 && col < m) {
      key = perm[col];
    } else if (col >= m && col < n) {
      key = col;
    }
  }

  swap(key, val, 1, get_bit(threadIdx.x, 1) ^ get_bit(threadIdx.x, 0));

  swap(key, val, 2, get_bit(threadIdx.x, 2) ^ get_bit(threadIdx.x, 1));
  swap(key, val, 1, get_bit(threadIdx.x, 2) ^ get_bit(threadIdx.x, 0));

  swap(key, val, 4, get_bit(threadIdx.x, 3) ^ get_bit(threadIdx.x, 2));
  swap(key, val, 2, get_bit(threadIdx.x, 3) ^ get_bit(threadIdx.x, 1));
  swap(key, val, 1, get_bit(threadIdx.x, 3) ^ get_bit(threadIdx.x, 0));

  swap(key, val, 8, get_bit(threadIdx.x, 4) ^ get_bit(threadIdx.x, 3));
  swap(key, val, 4, get_bit(threadIdx.x, 4) ^ get_bit(threadIdx.x, 2));
  swap(key, val, 2, get_bit(threadIdx.x, 4) ^ get_bit(threadIdx.x, 1));
  swap(key, val, 1, get_bit(threadIdx.x, 4) ^ get_bit(threadIdx.x, 0));

  swap(key, val, 16, get_bit(threadIdx.x, 4));
  swap(key, val, 8, get_bit(threadIdx.x, 3));
  swap(key, val, 4, get_bit(threadIdx.x, 2));
  swap(key, val, 2, get_bit(threadIdx.x, 1));
  swap(key, val, 1, get_bit(threadIdx.x, 0));

  if (threadIdx.x < nonzerosPerRow && row < m) {
    mtxIndL[idx] = (key == n) ? -1 : key;
    matrixValues[idx] = val;
  }
}

void PermuteColumns(SparseMatrix &A) {
  // Determine blocksize in x direction
  unsigned int dim_x = A.numberOfNonzerosPerRow;

  // Compute next power of two
  dim_x |= dim_x >> 1;
  dim_x |= dim_x >> 2;
  dim_x |= dim_x >> 4;
  dim_x |= dim_x >> 8;
  dim_x |= dim_x >> 16;
  ++dim_x;

  // Determine blocksize
  unsigned int dim_y = 512 / dim_x;

  // Compute next power of two
  dim_y |= dim_y >> 1;
  dim_y |= dim_y >> 2;
  dim_y |= dim_y >> 4;
  dim_y |= dim_y >> 8;
  dim_y |= dim_y >> 16;
  ++dim_y;

  // Shift right until we obtain a valid blocksize
  while (dim_x * dim_y > 512) {
    dim_y >>= 1;
  }

  // double *perm = new double[10];
  // double *vals = new double[10];
  // local_int_t *mtxIndl = new local_int_t[10];

  // hipMemcpy(perm, A.perm, sizeof(double) * 10, hipMemcpyDeviceToHost);
  // hipMemcpy(vals, A.d_matrixValues, sizeof(double) * 10,
  //            hipMemcpyDeviceToHost);
  // hipMemcpy(mtxIndl, A.d_mtxIndL, sizeof(local_int_t) * 10,
  //            hipMemcpyDeviceToHost);

  // printf("before launch perm\n");
  // for (int i = 0; i < 10; i++) {
  //   printf("perm, vals, mtxIndl [%d] : %f %f %d\n", i, perm[i], vals[i],
  //          mtxIndl[i]);
  // }

  if (dim_y == 32)
    LAUNCH_PERM_COLS(32, 32);
  else if (dim_y == 16)
    LAUNCH_PERM_COLS(32, 16);
  else if (dim_y == 8)
    LAUNCH_PERM_COLS(32, 8);
  else
    LAUNCH_PERM_COLS(32, 4);

  // hipMemcpy(perm, A.perm, sizeof(double) * 10, hipMemcpyDeviceToHost);
  // hipMemcpy(vals, A.d_matrixValues, sizeof(double) * 10,
  //            hipMemcpyDeviceToHost);
  // hipMemcpy(mtxIndl, A.d_mtxIndL, sizeof(local_int_t) * 10,
  //            hipMemcpyDeviceToHost);

  // // printf("after launch perm\n");
  // // for (int i = 0; i < 10; i++) {
  // //   printf("perm, vals, mtxIndl [%d] : %f %f %d\n", i, perm[i], vals[i],
  // //          mtxIndl[i]);
  // // }

  // free(perm);
  // free(vals);
  // free(mtxIndl);
}

void PermuteRows(SparseMatrix &A) {
  local_int_t m = A.localNumberOfRows;

  // Temporary structures for row permutation
  local_int_t *tmp_cols;
  double *tmp_vals;

  CUDA_CHECK_COMMAND(hipMalloc((void **)&tmp_cols, sizeof(local_int_t) * m));
  CUDA_CHECK_COMMAND(hipMalloc((void **)&tmp_vals, sizeof(double) * m));

  // Permute ELL rows
  for (local_int_t p = 0; p < A.ell_width; ++p) {
    local_int_t offset = p * m;

    CUDA_CHECK_COMMAND(hipMemcpy(tmp_cols, A.ell_col_ind + offset,
                                  sizeof(local_int_t) * m,
                                  hipMemcpyDeviceToDevice));
    CUDA_CHECK_COMMAND(hipMemcpy(tmp_vals, A.ell_val + offset,
                                  sizeof(double) * m,
                                  hipMemcpyDeviceToDevice));

    kernel_permute_ell_rows<1024><<<dim3((m - 1) / 1024 + 1), dim3(1024)>>>(
        m, p, tmp_cols, tmp_vals, A.perm, A.ell_col_ind, A.ell_val);
  }

  CUDA_CHECK_COMMAND(hipFree(tmp_cols));
  CUDA_CHECK_COMMAND(hipFree(tmp_vals));
}

template <unsigned int BLOCKSIZE>
__launch_bounds__(BLOCKSIZE) __global__
    void kernel_permute(local_int_t size, const local_int_t *perm,
                        const double *in, double *out) {
  local_int_t gid = blockIdx.x * BLOCKSIZE + threadIdx.x;

  if (gid >= size) {
    return;
  }

  out[perm[gid]] = in[gid];
}

void PermuteVector(local_int_t size, Vector &v, const local_int_t *perm) {
  double *buffer;
  CUDA_CHECK_COMMAND(
      hipMalloc((void **)&buffer, sizeof(double) * v.localLength));

  kernel_permute<1024><<<dim3((size - 1) / 1024 + 1), dim3(1024)>>>(
      size, perm, v.d_values, buffer);

  CUDA_CHECK_COMMAND(hipFree(v.d_values));
  v.d_values = buffer;
}
