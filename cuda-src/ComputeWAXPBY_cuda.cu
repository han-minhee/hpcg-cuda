#include "hip/hip_runtime.h"
#include "ComputeWAXPBY_cuda.cuh"
#include <ctime>

#define BLOCK_SIZE 32

__global__ void kernel_waxpby(local_int_t size, double alpha, const double *x,
                              double beta, const double *y, double *w) {
  local_int_t gid = blockIdx.x * blockDim.x + threadIdx.x;

  if (gid >= size) {
    return;
  }
  if (alpha == 1.0) {
    w[gid] = fma(beta, y[gid], x[gid]);
  } else if (beta == 1.0) {
    w[gid] = fma(alpha, x[gid], y[gid]);
  } else {
    w[gid] = fma(alpha, x[gid], beta * y[gid]);
  }
}

int ComputeWAXPBY_cuda_d(const local_int_t n, const double alpha,
                         const Vector &x, const double beta, const Vector &y,
                         Vector &w) {

  assert(x.localLength >= n);
  assert(y.localLength >= n);

  double *xv_d = x.d_values;
  double *yv_d = y.d_values;
  double *wv_d = w.d_values;

  assert(x.localLength >= n);
  assert(y.localLength >= n);
  assert(w.localLength >= n);

  size_t blockDim = (n - 1) / 512 + 1;
  size_t globalDim = 512;

  kernel_waxpby<<<blockDim, globalDim>>>(n, alpha, xv_d, beta, yv_d, wv_d);

  hipDeviceSynchronize();
  checkGPUandPrintLine;

  // size_t n_size = n * sizeof(double);
  // hipMemcpy(w.values, wv_d, n_size, hipMemcpyDeviceToHost);
  // checkGPUandPrintLine;
  // hipDeviceSynchronize();
  return 0;
}

int ComputeWAXPBY_cuda(const local_int_t n, const double alpha, const Vector &x,
                       const double beta, const Vector &y, Vector &w) {
  // if we remove memory operations,
  // the run speed will be 1/30
  // double begin = mytimer();
  assert(x.localLength >= n);
  assert(y.localLength >= n);

  double *xv = x.values;
  double *yv = y.values;
  double *wv = w.values;

  double *xv_d;
  double *yv_d;
  double *wv_d;
  assert(x.localLength >= n);
  assert(y.localLength >= n);
  assert(w.localLength >= n);

  size_t n_size = n * sizeof(double);
  printFileLine;

  hipMalloc((void **)&xv_d, n_size);
  checkGPUandPrintLine;

  hipMalloc((void **)&yv_d, n_size);
  checkGPUandPrintLine;

  hipMalloc((void **)&wv_d, n_size);
  checkGPUandPrintLine;

  hipMemcpy(xv_d, xv, n_size, hipMemcpyHostToDevice);
  checkGPUandPrintLine;

  hipMemcpy(yv_d, yv, n_size, hipMemcpyHostToDevice);
  checkGPUandPrintLine;

  // double endMemcpy = mytimer();

  size_t blockDim = (n - 1) / 512 + 1;
  size_t globalDim = 512;

  kernel_waxpby<<<blockDim, globalDim>>>(n, alpha, xv_d, beta, yv_d, wv_d);

  printFileLine;
  hipDeviceSynchronize();
  // double endGPU = mytimer();

  checkGPUandPrintLine;
  hipMemcpy(wv, wv_d, n_size, hipMemcpyDeviceToHost);
  checkGPUandPrintLine;
  hipDeviceSynchronize();
  hipFree(wv_d);
  hipFree(xv_d);
  hipFree(yv_d);

  // double endTotal = mytimer();

  // printf("GPU runtime : %f\n", endGPU - endMemcpy);
  // printf("total runtime : %f\n", endTotal - begin);

  return 0;
}

int ComputeWAXPBY_ref_cuda(const local_int_t n, const double alpha,
                           const Vector &x, const double beta, const Vector &y,
                           Vector &w) {

  assert(x.localLength >= n); // Test vector lengths
  assert(y.localLength >= n);

  const double *const xv = x.values;
  const double *const yv = y.values;
  double *const wv = w.values;

  if (alpha == 1.0) {
#ifndef HPCG_NO_OPENMP
#pragma omp parallel for
#endif
    for (local_int_t i = 0; i < n; i++)
      wv[i] = xv[i] + beta * yv[i];
  } else if (beta == 1.0) {
#ifndef HPCG_NO_OPENMP
#pragma omp parallel for
#endif
    for (local_int_t i = 0; i < n; i++)
      wv[i] = alpha * xv[i] + yv[i];
  } else {
#ifndef HPCG_NO_OPENMP
#pragma omp parallel for
#endif
    for (local_int_t i = 0; i < n; i++)
      wv[i] = alpha * xv[i] + beta * yv[i];
  }

  return 0;
}
