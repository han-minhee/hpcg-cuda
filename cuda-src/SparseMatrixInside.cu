#include "hip/hip_runtime.h"
#include "SparseMatrixInside.cuh"

#define LAUNCH_TO_ELL_COL(blocksizex, blocksizey)                              \
  kernel_to_ell_col<blocksizex, blocksizey>                                    \
      <<<dim3((A.localNumberOfRows - 1) / blocksizey + 1),                     \
         dim3(blocksizex, blocksizey)>>>(A.localNumberOfRows, A.ell_width,     \
                                         A.d_mtxIndL, A.ell_col_ind,           \
                                         d_halo_rows, A.halo_row_ind)

#define LAUNCH_TO_ELL_VAL(blocksizex, blocksizey)                              \
  kernel_to_ell_val<blocksizex, blocksizey>                                    \
      <<<dim3((A.localNumberOfRows - 1) / blocksizey + 1),                     \
         dim3(blocksizex, blocksizey)>>>(A.localNumberOfRows,                  \
                                         A.numberOfNonzerosPerRow,             \
                                         A.d_matrixValues, A.ell_val)

template <unsigned int BLOCKSIZE>
__launch_bounds__(BLOCKSIZE) __global__
    void kernel_copy_diagonal(local_int_t m, local_int_t n,
                              local_int_t ell_width,
                              const local_int_t *__restrict__ ell_col_ind,
                              const double *__restrict__ ell_val,
                              double *__restrict__ diagonal) {
  local_int_t row = blockIdx.x * BLOCKSIZE + threadIdx.x;

  if (row >= m) {
    return;
  }

  for (local_int_t p = 0; p < ell_width; ++p) {
    local_int_t idx = p * m + row;
    local_int_t col = ell_col_ind[idx];

    if (col >= 0 && col < n) {
      if (col == row) {
        diagonal[row] = ell_val[idx];
        break;
      }
    } else {
      break;
    }
  }
}

void CudaCopyMatrixDiagonalInside(const SparseMatrix &A, Vector &diagonal) {
  kernel_copy_diagonal<1024>
      <<<dim3((A.localNumberOfRows - 1) / 1024 + 1), dim3(1024)>>>(
          A.localNumberOfRows, A.localNumberOfColumns, A.ell_width,
          A.ell_col_ind, A.ell_val, diagonal.d_values);
}

template <unsigned int BLOCKSIZE>
__launch_bounds__(BLOCKSIZE) __global__
    void kernel_replace_diagonal(local_int_t m, local_int_t n,
                                 const double *__restrict__ diagonal,
                                 local_int_t ell_width,
                                 const local_int_t *__restrict__ ell_col_ind,
                                 double *__restrict__ ell_val,
                                 double *__restrict__ inv_diag) {
  local_int_t row = blockIdx.x * BLOCKSIZE + threadIdx.x;

  if (row >= m) {
    return;
  }

  double diag = diagonal[row];

  for (local_int_t p = 0; p < ell_width; ++p) {
    local_int_t idx = p * m + row;
    local_int_t col = ell_col_ind[idx];

    if (col >= 0 && col < n) {
      if (col == row) {
        ell_val[idx] = diag;
        break;
      }
    } else {
      break;
    }
  }

  inv_diag[row] = 1.0 / diag;
}

void CudaReplaceMatrixDiagonalInside(SparseMatrix &A, const Vector &diagonal) {
  kernel_replace_diagonal<1024>
      <<<dim3((A.localNumberOfRows - 1) / 1024 + 1), dim3(1024)>>>(
          A.localNumberOfRows, A.localNumberOfColumns, diagonal.d_values,
          A.ell_width, A.ell_col_ind, A.ell_val, A.inv_diag);
}

template <unsigned int BLOCKSIZEX, unsigned int BLOCKSIZEY>
__launch_bounds__(BLOCKSIZEX *BLOCKSIZEY) __global__
    void kernel_to_ell_col(local_int_t m, local_int_t nonzerosPerRow,
                           const local_int_t *__restrict__ mtxIndL,
                           local_int_t *__restrict__ ell_col_ind,
                           local_int_t *__restrict__ halo_rows,
                           local_int_t *__restrict__ halo_row_ind) {
  local_int_t row = blockIdx.x * BLOCKSIZEY + threadIdx.y;

#ifndef HPCG_NO_MPI
  __shared__ bool sdata[BLOCKSIZEY];
  sdata[threadIdx.y] = false;
  hipDeviceSynchronize();
  __synchtreads();
#endif

  if (row >= m) {
    return;
  }

  local_int_t col = __ldg(mtxIndL + row * nonzerosPerRow + threadIdx.x);
  ell_col_ind[threadIdx.x * m + row] = col;

#ifndef HPCG_NO_MPI
  if (col >= m) {
    sdata[threadIdx.y] = true;
  }

  __syncthreads();

  if (threadIdx.x == 0) {
    if (sdata[threadIdx.y] == true) {
      halo_row_ind[atomicAdd(halo_rows, 1)] = row;
    }
  }
#endif
}

template <unsigned int BLOCKSIZEX, unsigned int BLOCKSIZEY>
__launch_bounds__(BLOCKSIZEX *BLOCKSIZEY) __global__
    void kernel_to_ell_val(local_int_t m, local_int_t nnz_per_row,
                           const double *__restrict__ matrixValues,
                           double *__restrict__ ell_val) {
  local_int_t row = blockIdx.x * BLOCKSIZEY + threadIdx.y;

  if (row >= m) {
    return;
  }

  local_int_t idx = threadIdx.x * m + row;
  ell_val[idx] = matrixValues[row * nnz_per_row + threadIdx.x];
}

template <unsigned int BLOCKSIZE>
__launch_bounds__(BLOCKSIZE) __global__
    void kernel_to_halo(local_int_t halo_rows, local_int_t m, local_int_t n,
                        local_int_t ell_width,
                        const local_int_t *__restrict__ ell_col_ind,
                        const double *__restrict__ ell_val,
                        const local_int_t *__restrict__ halo_row_ind,
                        local_int_t *__restrict__ halo_col_ind,
                        double *__restrict__ halo_val) {
  local_int_t gid = blockIdx.x * BLOCKSIZE + threadIdx.x;

  if (gid >= halo_rows) {
    return;
  }

  local_int_t row = halo_row_ind[gid];

  int q = 0;
  for (int p = 0; p < ell_width; ++p) {
    local_int_t ell_idx = p * m + row;
    local_int_t col = ell_col_ind[ell_idx];

    if (col >= m && col < n) {
      local_int_t halo_idx = q++ * halo_rows + gid;

      halo_col_ind[halo_idx] = col;
      halo_val[halo_idx] = ell_val[ell_idx];
    }
  }

  for (; q < ell_width; ++q) {
    local_int_t idx = q * halo_rows + gid;
    halo_col_ind[idx] = -1;
  }
}

void ConvertToELLInside(SparseMatrix &A) {
  // We can re-use mtxIndL array for ELL values
  A.ell_val = reinterpret_cast<double *>(A.d_mtxIndG);
  A.d_mtxIndG = NULL;

  // Resize
  // TODO: implement cudaRealloc
  double *tempRealloc;
  cudaHostRealloc(A.ell_val, tempRealloc,
                  sizeof(double) * A.ell_width * A.localNumberOfRows);

  // Determine blocksize
  unsigned int blocksize = 1024 / A.ell_width;

  // Compute next power of two
  blocksize |= blocksize >> 1;
  blocksize |= blocksize >> 2;
  blocksize |= blocksize >> 4;
  blocksize |= blocksize >> 8;
  blocksize |= blocksize >> 16;
  ++blocksize;

  // Shift right until we obtain a valid blocksize
  while (blocksize * A.ell_width > 1024) {
    blocksize >>= 1;
  }

  if (blocksize == 32)
    LAUNCH_TO_ELL_VAL(27, 32);
  else if (blocksize == 16)
    LAUNCH_TO_ELL_VAL(27, 16);
  else if (blocksize == 8)
    LAUNCH_TO_ELL_VAL(27, 8);
  else
    LAUNCH_TO_ELL_VAL(27, 4);

  // We can re-use mtxIndG array for the ELL column indices
  A.ell_col_ind = reinterpret_cast<local_int_t *>(A.d_matrixValues);
  A.d_matrixValues = NULL;

  // Resize the array

  cudaHostRealloc(A.ell_col_ind, tempRealloc,
                  sizeof(local_int_t) * A.ell_width * A.localNumberOfRows);

  // Convert mtxIndL into ELL column indices
  local_int_t *d_halo_rows = reinterpret_cast<local_int_t *>(workspace);

#ifndef HPCG_NO_MPI
  CUDA_CHECK_COMMAND(hipMalloc((void **)&A.halo_row_ind,
                                sizeof(local_int_t) * A.totalToBeSent));

  CUDA_CHECK_COMMAND(hipMemset(d_halo_rows, 0, sizeof(local_int_t)));
#endif

  if (blocksize == 32)
    LAUNCH_TO_ELL_COL(27, 32);
  else if (blocksize == 16)
    LAUNCH_TO_ELL_COL(27, 16);
  else if (blocksize == 8)
    LAUNCH_TO_ELL_COL(27, 8);
  else
    LAUNCH_TO_ELL_COL(27, 4);

  // Free old matrix indices
  CUDA_CHECK_COMMAND(hipFree(A.d_mtxIndL));

#ifndef HPCG_NO_MPI
  CUDA_CHECK_COMMAND(hipMemcpy(&A.halo_rows, d_halo_rows, sizeof(local_int_t),
                                hipMemcpyDeviceToHost));
  assert(A.halo_rows <= A.totalToBeSent);

  CUDA_CHECK_COMMAND(
      hipMalloc((void **)&A.halo_col_ind,
                 sizeof(local_int_t) * A.ell_width * A.halo_rows));
  CUDA_CHECK_COMMAND(hipMalloc((void **)&A.halo_val,
                                sizeof(double) * A.ell_width * A.halo_rows));

  size_t rocprim_size;
  void *rocprim_buffer = NULL;

  // TODO: FIX radix sort
  CUDA_CHECK_COMMAND(hipcub::DeviceRadixSort::SortKeys(
      rocprim_buffer, rocprim_size, A.halo_row_ind, A.halo_row_ind,
      A.halo_rows));
  CUDA_CHECK_COMMAND(hipMalloc(&rocprim_buffer, rocprim_size));
  CUDA_CHECK_COMMAND(hipcub::DeviceRadixSort::SortKeys(
      rocprim_buffer, rocprim_size, A.halo_row_ind,
      A.halo_row_ind, // TODO inplace!
      A.halo_rows));
  CUDA_CHECK_COMMAND(hipFree(rocprim_buffer));

  kernel_to_halo<128><<<dim3((A.halo_rows - 1) / 128 + 1), dim3(128)>>>(
      A.halo_rows, A.localNumberOfRows, A.localNumberOfColumns, A.ell_width,
      A.ell_col_ind, A.ell_val, A.halo_row_ind, A.halo_col_ind, A.halo_val);
#endif
}

template <unsigned int BLOCKSIZE>
__launch_bounds__(BLOCKSIZE) __global__
    void kernel_extract_diag_index(local_int_t m, local_int_t ell_width,
                                   const local_int_t *__restrict__ ell_col_ind,
                                   const double *__restrict__ ell_val,
                                   local_int_t *__restrict__ diag_idx,
                                   double *__restrict__ inv_diag) {
  local_int_t row = blockIdx.x * BLOCKSIZE + threadIdx.x;

  if (row >= m) {
    return;
  }

  for (local_int_t p = 0; p < ell_width; ++p) {
    local_int_t idx = p * m + row;
    local_int_t col = ell_col_ind[idx];

    if (col == row) {
      diag_idx[row] = p;
      inv_diag[row] = 1.0 / ell_val[idx];
      break;
    }
  }
}

void ExtractDiagonalInside(SparseMatrix &A) {
  local_int_t m = A.localNumberOfRows;
  // Allocate memory to extract diagonal entries
  CUDA_CHECK_COMMAND(hipMalloc((void **)&A.diag_idx, sizeof(local_int_t) * m));
  CUDA_CHECK_COMMAND(hipMalloc((void **)&A.inv_diag, sizeof(double) * m));
  // Extract diagonal entries
  kernel_extract_diag_index<1024><<<dim3((m - 1) / 1024 + 1), dim3(1024)>>>(
      m, A.ell_width, A.ell_col_ind, A.ell_val, A.diag_idx, A.inv_diag);
}
