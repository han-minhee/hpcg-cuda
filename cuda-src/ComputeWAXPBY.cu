#include "hip/hip_runtime.h"

 #include "ComputeWAXPBY_cuda.hpp"

 #include <cassert>
 /*!
   Routine to compute the update of a vector with the sum of two
   scaled vectors where: w = alpha*x + beta*y
 
   This is the reference WAXPBY impmentation.  It CANNOT be modified for the
   purposes of this benchmark.
 
   @param[in] n the number of vector elements (on this processor)
   @param[in] alpha, beta the scalars applied to x and y respectively.
   @param[in] x, y the input vectors
   @param[out] w the output vector.
 
   @return returns 0 upon success and non-zero otherwise
 
   @see ComputeWAXPBY
 */


// hipblasDcopy
/*

hipblasStatus_t hipblasDcopy(hipblasHandle_t handle, int n,
                           const double          *x, int incx,
                           double                *y, int incy)

hipblasStatus_t  hipblasDscal(hipblasHandle_t handle, int n,
                            const double          *alpha,
                            double          *x, int incx)
                            
hipblasStatus_t hipblasDaxpy(hipblasHandle_t handle, int n,
                           const double          *alpha,
                           const double          *x, int incx,
                           double                *y, int incy)

*/
// hipblasDaxpy

__global__ void kernelWAXPBY(int n
		  double alpha,
		  double beta,
		  double *  xv,
		  double *  yv,
		  double *  wv
		)               
{
    int global_index;
    int local_index;
	for(local_index = global_index; local_index < /* FIXME : limits*/; local_index++){
			if (alpha==1.0) {
			    wv[x] = xv[local_index] + beta * yv[local_index];
			} else if (beta==1.0) {
			    wv[x] = alpha * xv[local_index] + yv[local_index];
			} else  {
			    wv[x] = alpha * xv[local_index] + beta * yv[local_index];
			}
			
	}
}

 int ComputeWAXPBY_cuda_0(const local_int_t n, const double alpha, const Vector & x,
     const double beta, const Vector & y, Vector & w) {
 
        // Test vector lengths
        // should also be in the kernel?
   assert(x.localLength>=n); 
   assert(y.localLength>=n);
 
   const double * const xv = x.values;
   const double * const yv = y.values;
   double * const wv = w.values;

   //global dim : n
   //local dim : for now, just 1?

   /*
   if (alpha==1.0) {
     for (local_int_t i=0; i<n; i++) wv[i] = xv[i] + beta * yv[i];
   } else if (beta==1.0) {
     for (local_int_t i=0; i<n; i++) wv[i] = alpha * xv[i] + yv[i];
   } else  {
     for (local_int_t i=0; i<n; i++) wv[i] = alpha * xv[i] + beta * yv[i];
   }
*/ 
   return 0;
 }
 