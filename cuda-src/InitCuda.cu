#include "InitCuda.cuh"
// macro ref:
// https://stackoverflow.com/questions/14038589/what-is-the-canonical-way-to-check-for-errors-using-the-cuda-runtime-api

int gpuAssert(hipError_t code) {
    if (code != hipSuccess) {
      printf("GPUassert: %s\n", hipGetErrorString(code));
      return -1;
    } else {
      return 0;
    }
  }

int getDeviceProp(hipDeviceProp_t deviceProp) {
  if (hipSuccess != hipGetDeviceProperties(&deviceProp, 0)) {
    return 0;
  } else {
    return 1;
  }
}

int initDevice(void) {
  hipDeviceProp_t deviceProp;

  int getDevicePropSuccess = getDeviceProp(deviceProp);
  if (!getDevicePropSuccess) {
    return -1;
  }

  deviceWarpSize = deviceProp.warpSize;
  regsPerBlock = deviceProp.regsPerBlock;
  multiProcessorCount = deviceProp.multiProcessorCount;
  return 0;
}