
#include "../src/hpcg.hpp"
#include "finalizeInside.cuh"
#include <fstream>

/*!
  Closes the I/O stream used for logging information throughout the HPCG run.

  @return returns 0 upon success and non-zero otherwise

  @see HPCG_Init
*/
int HPCG_FinalizeInside(void) {
  HPCG_fout.close();

  // Destroy streams
  CUDA_CHECK_COMMAND(hipStreamDestroy(stream_interior));
  CUDA_CHECK_COMMAND(hipStreamDestroy(stream_halo));

  // Free workspace
  CUDA_CHECK_COMMAND(hipFree(workspace));

#ifdef HPCG_MEMMGMT
  // Clear allocator
  CUDA_CHECK_COMMAND(allocator.Clear());
#endif

  // Reset HIP device
  hipDeviceReset();

  return 0;
}
