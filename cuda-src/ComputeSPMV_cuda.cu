#include "hip/hip_runtime.h"
#include "ComputeSPMV_cuda.cuh"
__global__ void kernelSPMV(int n, double *x, double *y, int localNumberOfRows,
                           double **matrixValues, int **mtxIndL,
                           int **nonzerosInRow) {}

int ComputeSPMV_cuda_cusparse(const SparseMatrix &A, Vector &x, Vector &y) {
  assert(x.localLength >= A.localNumberOfColumns);
  assert(y.localLength >= A.localNumberOfRows);

  double *xv = x.values;
  double *yv = y.values;

  double *xv_d;
  double *yv_d;

  size_t M = A.localNumberOfRows;
  size_t N = A.localNumberOfColumns;

  double *flatMatrixVal = new double[M * 27];
  int *flatColIndices = new int[M * 27];
  int *flatRowOffsets = new int[M + 1];

  clock_t start;

  start = clock();
  #pragma omp parallel for
  for (int i = 0; i < M; i++) {
    for (int j = 0; j < 27; j++) {
      flatMatrixVal[i * 27 + j] = A.matrixValues[i][j];
      flatColIndices[i * 27 + j] = A.mtxIndL[i][j];
    }
    flatRowOffsets[i] = 27 * i;
  }

  flatRowOffsets[M] = flatRowOffsets[M - 1] + 1;

  hipMalloc((void **)&xv_d, N * sizeof(double));
  if (gpuCheckError() == -1) {
    return -1;
  }

  hipMalloc((void **)&yv_d, M * sizeof(double));
  if (gpuCheckError() == -1) {
    return -1;
  }

  hipMemcpy(xv_d, xv, N * sizeof(double), hipMemcpyHostToDevice);
  if (gpuCheckError() == -1) {
    return -1;
  }
  
  hipDeviceSynchronize();
  // --- Device side dense matrix

  hipsparseHandle_t handle;
  hipsparseCreate(&handle);

  if (gpuCheckError() == -1) {
    return -1;
  }

  hipsparseDnVecDescr_t vecX, vecY;
  hipsparseCreateDnVec(&vecX, N, xv_d, HIP_R_64F);
  hipsparseCreateDnVec(&vecY, M, yv_d, HIP_R_64F);

  if (gpuCheckError() == -1) {
    return -1;
  }

  int *csrRowOffsets_d;
  int *csrColInd_d;
  double *csrValues_d;

  hipMalloc((void **)&csrRowOffsets_d, (M + 1) * sizeof(int));
  if (gpuCheckError() == -1) {

    return -1;
  }
  // printf("line passed %d\n", __LINE__);

  hipMalloc((void **)&csrColInd_d, (M * 27) * sizeof(int));
  if (gpuCheckError() == -1) {

    return -1;
  }
  // printf("line passed %d\n", __LINE__);

  hipMalloc((void **)&csrValues_d, (M * 27) * sizeof(double));
  if (gpuCheckError() == -1) {

    return -1;
  }
  // printf("line passed %d\n", __LINE__);

  hipMemcpy(csrColInd_d, flatColIndices, (M * 27) * sizeof(int),
             hipMemcpyHostToDevice);
  if (gpuCheckError() == -1) {

    return -1;
  }
  // printf("line passed %d\n", __LINE__);

  hipMemcpy(csrRowOffsets_d, flatRowOffsets, (M + 1) * sizeof(int),
             hipMemcpyHostToDevice);
  if (gpuCheckError() == -1) {

    return -1;
  }
  // printf("line passed %d\n", __LINE__);

  hipMemcpy(csrValues_d, flatMatrixVal, (M * 27) * sizeof(double),
             hipMemcpyHostToDevice);
  if (gpuCheckError() == -1) {

    return -1;
  }
  // printf("line passed %d\n", __LINE__);

  hipsparseSpMatDescr_t matA;
  hipsparseCreateCsr(&matA, M, N, M * 27, csrRowOffsets_d, csrColInd_d,
                    csrValues_d, HIPSPARSE_INDEX_32I, HIPSPARSE_INDEX_32I,
                    HIPSPARSE_INDEX_BASE_ZERO, HIP_R_64F);
  if (gpuCheckError() == -1) {

    return -1;
  }
  const double alpha = 1.0;
  const double beta = 0.0;

  size_t bufferSize;
  double *buffer_d;

  hipsparseSpMV_bufferSize(handle, HIPSPARSE_OPERATION_NON_TRANSPOSE, &alpha,
                          matA, vecX, &beta, vecY, HIP_R_64F,
                          HIPSPARSE_CSRMV_ALG1, &bufferSize);
  if (gpuCheckError() == -1) {

    return -1;
  }
  // printf("line passed %d\n", __LINE__);

  hipMalloc((void **)&buffer_d, bufferSize);
  if (gpuCheckError() == -1) {
    return -1;
  }
  // printf("line passed %d\n", __LINE__);

  hipsparseSpMV(handle, HIPSPARSE_OPERATION_NON_TRANSPOSE, &alpha, matA, vecX,
               &beta, vecY, HIP_R_64F, HIPSPARSE_CSRMV_ALG1, buffer_d);

  hipMemcpy(yv, yv_d, M * sizeof(double), hipMemcpyDeviceToHost);
  if (gpuCheckError() == -1) {
    return -1;
  }
  // printf("line passed %d\n", __LINE__);

  hipFree(xv_d);
  hipFree(yv_d);
  hipFree(buffer_d);
  hipFree(csrRowOffsets_d);
  hipFree(csrColInd_d);
  hipFree(csrValues_d);
  hipsparseDestroySpMat(matA);
  hipsparseDestroyDnVec(vecX);
  hipsparseDestroyDnVec(vecY);
  hipsparseDestroy(handle);
  free(flatColIndices);
  free(flatMatrixVal);
  free(flatRowOffsets);

  return 0;
}
int ComputeSPMV_ref_cuda(const SparseMatrix &A, Vector &x, Vector &y) {
  assert(x.localLength >= A.localNumberOfColumns); // Test vector lengths
  assert(y.localLength >= A.localNumberOfRows);

  //*const : can only alter the object being pointed
  // const double *const: cannot alter pointer and the object

  const double *const xv = x.values;
  double *const yv = y.values;

  const local_int_t nrow = A.localNumberOfRows;
  for (local_int_t i = 0; i < nrow; i++) {
    double sum = 0.0;
    const double *const cur_vals = A.matrixValues[i];
    const local_int_t *const cur_inds = A.mtxIndL[i];
    const int cur_nnz = A.nonzerosInRow[i];

    for (int j = 0; j < cur_nnz; j++)
      sum += cur_vals[j] * xv[cur_inds[j]];
    yv[i] = sum;
  }
  return 0;
}
int ComputeSPMV_cuda(const SparseMatrix &A, Vector &x, Vector &y) {

}
